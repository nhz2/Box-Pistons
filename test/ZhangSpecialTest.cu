#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include "catch.hpp"
#include <hip/hip_runtime.h>
#include <cuda_utils.h>
#include <Box-Pistons-Orthogonal/ZhangSpecial.h>

template<typename T, int N>
struct H_D{
    T* h;
    T* d;
    H_D(){
        h= (T*)malloc(N*sizeof(T));
        hipMalloc((void**)&d,N*sizeof(T));
    }
    void c2d(){
        cudaCheck(hipMemcpy(d,h,N*sizeof(T),hipMemcpyHostToDevice));
    }
    void c2h(){
        cudaCheck(hipMemcpy(h,d,N*sizeof(T),hipMemcpyDeviceToHost));
    }
};


TEST_CASE("Constant pressure piston drift"){
    H_D<double3,1> virial;
    H_D<double3,1> comke;
    H_D<double3,1> box;
    H_D<double3,1> box_dot;
    H_D<double3,1> v_prescale;
    H_D<double3,1> v_scale;
    H_D<double3,1> r_prescale;
    //initialize everything to zero
    double3 zero3;
    zero3.x=0;
    zero3.y=0;
    zero3.z=0;
    double3 one3;
    one3.x=1;
    one3.y=1;
    one3.z=1;
    boxpiston::ZhangSpecial piston;
    piston.xgamma= 0;
    piston.zgamma= 0;
    piston.xpiston_mass= 10;
    piston.zpiston_mass= 10;
    piston.pressure=1;
    piston.surface_tension=0;
    piston.k= 1;
    *virial.h= zero3;
    *comke.h= one3;
    *box.h= one3;
    *box_dot.h= zero3;
    double timestep= 0.01;
    SECTION("NPH ideal gas simulation"){
        double initial_total_energy= piston.pe(*box.h)+piston.ke(*box.h,*box_dot.h)+
                            comke.h->x+comke.h->y+comke.h->z;
        for (int i=0;i<1000000;i++){
            piston.virialKick(*virial.h,*box.h,*box_dot.h,*box_dot.h,timestep*0.5);
            piston.drift(*comke.h,*box.h,*box_dot.h,*comke.h,*box.h,*box_dot.h,
              *v_scale.h,*v_prescale.h,*r_prescale.h,timestep*0.5);
            piston.thermostat(*box.h,*box_dot.h,*box_dot.h,85,timestep,0,0,i*256);
            piston.drift(*comke.h,*box.h,*box_dot.h,*comke.h,*box.h,*box_dot.h,
              *v_scale.h,*v_prescale.h,*r_prescale.h,timestep*0.5);
            piston.virialKick(*virial.h,*box.h,*box_dot.h,*box_dot.h,timestep*0.5);
            double final_total_energy= piston.pe(*box.h)+piston.ke(*box.h,*box_dot.h)+
                                comke.h->x+comke.h->y+comke.h->z;
            REQUIRE(final_total_energy==Approx(initial_total_energy).epsilon(1E-4));
        }
        piston.pressure=5;
        piston.surface_tension=1;
        initial_total_energy= piston.pe(*box.h)+piston.ke(*box.h,*box_dot.h)+
                            comke.h->x+comke.h->y+comke.h->z;
        for (int i=0;i<1000000;i++){
          piston.virialKick(*virial.h,*box.h,*box_dot.h,*box_dot.h,timestep*0.5);
          piston.drift(*comke.h,*box.h,*box_dot.h,*comke.h,*box.h,*box_dot.h,
            *v_scale.h,*v_prescale.h,*r_prescale.h,timestep*0.5);
          piston.thermostat(*box.h,*box_dot.h,*box_dot.h,85,timestep,0,1,i*256);
          piston.drift(*comke.h,*box.h,*box_dot.h,*comke.h,*box.h,*box_dot.h,
            *v_scale.h,*v_prescale.h,*r_prescale.h,timestep*0.5);
          piston.virialKick(*virial.h,*box.h,*box_dot.h,*box_dot.h,timestep*0.5);
          double final_total_energy= piston.pe(*box.h)+piston.ke(*box.h,*box_dot.h)+
                              comke.h->x+comke.h->y+comke.h->z;
          REQUIRE(final_total_energy==Approx(initial_total_energy).epsilon(1E-4));
        }
    }
}
